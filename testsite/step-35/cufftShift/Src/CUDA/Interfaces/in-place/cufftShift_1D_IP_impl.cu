/*********************************************************************
 * Copyright © 2011-2014,
 * Marwan Abdellah: <abdellah.marwan@gmail.com>
 *
 * This library (cufftShift) is free software; you can redistribute it
 * and/or modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 ********************************************************************/

#ifndef CUFFTSHIFT_1D_SINGLE_ARRAY_IMPL_CU
#define CUFFTSHIFT_1D_SINGLE_ARRAY_IMPL_CU

#include <cuUtils/configGPU.h>
#include "cufftShiftShared.h"
#include <CUDA/Kernels/in-place/cufftShift_1D_IP.cu>

template <typename T>
extern
void cufftShift_1D_impl(T* data, int NX)
{
    kernelConf* conf = cufftShift::GenAutoConf_1D(NX/2);
    cufftShift_1D_kernel <<< conf->grid, conf->block >>> (data, NX);
}

template <typename T>
extern
void cufftShift_1D_config_impl(T* data, int NX, kernelConf* conf)
{
    cufftShift_1D_kernel <<< conf->grid, conf->block >>> (data, NX);
}

template void cufftShift_1D_impl <hipfftReal>
(hipfftReal* data, int NX);

template void cufftShift_1D_impl <hipfftDoubleReal>
(hipfftDoubleReal* data, int NX);

template void cufftShift_1D_impl <hipfftComplex>
(hipfftComplex* data, int NX);

template void cufftShift_1D_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* data, int NX);

template void cufftShift_1D_config_impl <hipfftReal>
(hipfftReal* data, int NX, kernelConf* conf);

template void cufftShift_1D_config_impl <hipfftDoubleReal>
(hipfftDoubleReal* data, int NX, kernelConf* conf);

template void cufftShift_1D_config_impl <hipfftComplex>
(hipfftComplex* data, int NX, kernelConf* conf);

template void cufftShift_1D_config_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* data, int NX, kernelConf* conf);

#endif // CUFFTSHIFT_1D_SINGLE_ARRAY_IMPL_CU
