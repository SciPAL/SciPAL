/*********************************************************************
 * Copyright © 2011-2014,
 * Marwan Abdellah: <abdellah.marwan@gmail.com>
 *
 * This library (cufftShift) is free software; you can redistribute it
 * and/or modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 ********************************************************************/

#ifndef CUFFTSHIFT_2D_SINGLE_ARRAY_IMPL_CU
#define CUFFTSHIFT_2D_SINGLE_ARRAY_IMPL_CU

#include <cuUtils/configGPU.h>
#include "cufftShiftShared.h"
#include <CUDA/Kernels/in-place/cufftShift_2D_IP.cu>

template <typename T>
extern
void cufftShift_2D_impl(T* data, int NX, int NY)
{
    if (NX == NY)
    {
        const int N = NX;
        kernelConf* conf = cufftShift::GenAutoConf_2D(N/2);
        cufftShift_2D_kernel <<< conf->grid, conf->block >>> (data, N);
    }
    else
    {
        printf("The library is supporting NxN arrays only \n");
        exit(0);
    }
    hipDeviceSynchronize();
}

template <typename T>
extern
void cufftShift_2D_config_impl(T* data, int NX, int NY, kernelConf* conf)
{
    if (NX == NY)
    {
        const int N = NX;
        cufftShift_2D_kernel <<< conf->grid, conf->block >>> (data, N);
    }

    else
    {
        printf("The library is supporting NxN arrays only \n");
        exit(0);
    }
}

template void cufftShift_2D_impl <hipfftReal>
(hipfftReal* data, int NX, int NY);

template void cufftShift_2D_impl <hipfftDoubleReal>
(hipfftDoubleReal* data, int NX, int NY);

template void cufftShift_2D_impl <hipfftComplex>
(hipfftComplex* data, int NX, int NY);

template void cufftShift_2D_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* data, int NX, int NY);

template void cufftShift_2D_config_impl <hipfftReal>
(hipfftReal* data, int NX, int NY, kernelConf* conf);

template void cufftShift_2D_config_impl <hipfftDoubleReal>
(hipfftDoubleReal* data, int NX, int NY, kernelConf* conf);

template void cufftShift_2D_config_impl <hipfftComplex>
(hipfftComplex* data, int NX, int NY, kernelConf* conf);

template void cufftShift_2D_config_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* data, int NX, int NY, kernelConf* conf);

#endif // CUFFTSHIFT_2D_SINGLE_ARRAY_IMPL_CU
