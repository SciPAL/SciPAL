/*********************************************************************
 * Copyright © 2011-2014,
 * Marwan Abdellah: <abdellah.marwan@gmail.com>
 *
 * This library (cufftShift) is free software; you can redistribute it
 * and/or modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 ********************************************************************/

#ifndef CUFFTSHIFT_2D_IMPL_CU
#define CUFFTSHIFT_2D_IMPL_CU

#include <cuUtils/configGPU.h>
#include "cufftShiftShared.h"
#include <CUDA/Kernels/out-of-place/cufftShift_2D_OP.cu>

template <typename T>
extern
void cufftShift_2D_impl(T* input, T* output, int NX, int NY)
{
    if (NX == NY)
    {
        const int N = NX;
        kernelConf* conf = cufftShift::GenAutoConf_2D(N);
        cufftShift_2D_kernel <<< conf->grid, conf->block >>> (input, output, N);
    }
    else
    {
        printf("The library is supporting NxN arrays only \n");
        exit(0);
    }
}

template <typename T>
extern
void cufftShift_2D_config_impl(T* input, T* output, int NX, int NY, kernelConf* conf)
{
    if (NX == NY)
    {
        const int N = NX;
        cufftShift_2D_kernel <<< conf->grid, conf->block >>> (input, output, N);
    }

    else
    {
        printf("The library is supporting NxN arrays only \n");
        exit(0);
    }
}

template void cufftShift_2D_impl <hipfftReal>
(hipfftReal* input, hipfftReal* output, int NX, int NY);

template void cufftShift_2D_impl <hipfftDoubleReal>
(hipfftDoubleReal* input, hipfftDoubleReal* output, int NX, int NY);

template void cufftShift_2D_impl <hipfftComplex>
(hipfftComplex* input, hipfftComplex* output, int NX, int NY);

template void cufftShift_2D_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* input, hipfftDoubleComplex* output, int NX, int NY);

template void cufftShift_2D_config_impl <hipfftReal>
(hipfftReal* input, hipfftReal* output, int NX, int NY, kernelConf* conf);

template void cufftShift_2D_config_impl <hipfftDoubleReal>
(hipfftDoubleReal* input, hipfftDoubleReal* output, int NX, int NY, kernelConf* conf);

template void cufftShift_2D_config_impl <hipfftComplex>
(hipfftComplex* input, hipfftComplex* output, int NX, int NY, kernelConf* conf);

template void cufftShift_2D_config_impl <hipfftDoubleComplex>
(hipfftDoubleComplex* input, hipfftDoubleComplex* output, int NX, int NY, kernelConf* conf);

#endif // CUFFTSHIFT_2D_IMPL_CU
