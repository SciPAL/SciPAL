/*********************************************************************
 * Copyright © 2011-2014,
 * Marwan Abdellah: <abdellah.marwan@gmail.com>
 *
 * This library (cufftShift) is free software; you can redistribute it
 * and/or modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 ********************************************************************/
 
#ifndef CUFFTSHIFT_1D_SINGLE_ARRAY_CU
#define CUFFTSHIFT_1D_SINGLE_ARRAY_CU

#include <hip/hip_runtime.h>

template <typename T>
__global__
void cufftShift_1D_kernel(T* data, int NX)
{
    int threadIdxX = threadIdx.x;
    int blockDimX = blockDim.x;
    int blockIdxX = blockIdx.x;

    int index = ((blockIdxX * blockDimX) + threadIdxX);
    if (index < NX/2)
    {
        // Save the first value
        T regTemp = data[index];

        // Swap the first element
        data[index] = (T) data[index + (NX / 2)];

        // Swap the second one
        data[index + (NX / 2)] = (T) regTemp;
    }
}

#endif // CUFFTSHIFT_1D_SINGLE_ARRAY_CU
