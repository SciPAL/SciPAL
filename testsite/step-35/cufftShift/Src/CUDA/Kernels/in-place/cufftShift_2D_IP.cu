/*********************************************************************
 * Copyright © 2011-2014,
 * Marwan Abdellah: <abdellah.marwan@gmail.com>
 *
 * This library (cufftShift) is free software; you can redistribute it
 * and/or modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 ********************************************************************/

#ifndef CUFFTSHIFT_2D_SINGLE_ARRAY_CU
#define CUFFTSHIFT_2D_SINGLE_ARRAY_CU

#include <hip/hip_runtime.h>
#include <stdio.h>

template <typename T>
__global__
void cufftShift_2D_kernel(T* data, int N)
{
    // 2D Slice & 1D Line
    const int sLine = N;
    const int sSlice = N * N;

    // Transformations Equations
    int sEq1 = (sSlice + sLine) / 2;
    int sEq2 = (sSlice - sLine) / 2;

    // Thread Index (1D)
    int xThreadIdx = threadIdx.x;
    int yThreadIdx = threadIdx.y;

    // Block Width & Height
    int blockWidth = blockDim.x;
    int blockHeight = blockDim.y;

    // Thread Index (2D)
    int xIndex = blockIdx.x * blockWidth + xThreadIdx;
    int yIndex = blockIdx.y * blockHeight + yThreadIdx;

    // Thread Index Converted into 1D Index
    int index = (yIndex * N) + xIndex;

    T regTemp;

//    if (xIndex < N / 2)
//    {
//        if (yIndex < N / 2)
//        {
            regTemp = data[index];

            // First Quad
            data[index] = data[index + sEq1];

            // Third Quad
            data[index + sEq1] = regTemp;
//        }
//    }
//    else
//    {
//        if (yIndex >= N / 2)
//        {
            index += sLine/2;
//            printf("peng\n");
            regTemp = data[index];

            // Second Quad
            data[index] = data[index + sEq2];

            // Fourth Quad
            data[index + sEq2] = regTemp;
//        }
//    }
}

#endif // CUFFTSHIFT_2D_SINGLE_ARRAY_CU
