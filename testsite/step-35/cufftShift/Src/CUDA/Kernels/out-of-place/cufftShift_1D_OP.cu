/*********************************************************************
 * Copyright © 2011-2014,
 * Marwan Abdellah: <abdellah.marwan@gmail.com>
 *
 * This library (cufftShift) is free software; you can redistribute it
 * and/or modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 ********************************************************************/
 
#ifndef CUFFTSHIFT_1D_CU
#define CUFFTSHIFT_1D_CU

#include <hip/hip_runtime.h>

template <typename T>
__global__
void cufftShift_1D_kernel(T* input, T* output, int NX)
{
    int threadIdxX = threadIdx.x;
    int blockDimX = blockDim.x;
    int blockIdxX = blockIdx.x;

    int index = ((blockIdxX * blockDimX) + threadIdxX);

    if (index < (NX / 2))
        output[index] = (T) input[index + (NX / 2)];
    else
        output[index] = (T) input[index - (NX / 2)];
}


#endif // CUFFTSHIFT_1D_CU
