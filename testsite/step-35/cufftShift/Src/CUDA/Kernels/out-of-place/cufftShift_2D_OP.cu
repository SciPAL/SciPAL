/*********************************************************************
 * Copyright © 2011-2014,
 * Marwan Abdellah: <abdellah.marwan@gmail.com>
 *
 * This library (cufftShift) is free software; you can redistribute it
 * and/or modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 ********************************************************************/
 
#ifndef CUFFTSHIFT_2D_CU
#define CUFFTSHIFT_2D_CU

#include <hip/hip_runtime.h>

template <typename T>
__global__
void cufftShift_2D_kernel(T* input, T* output, int N)
{
    // 2D Slice & 1D Line
    int sLine = N;
    int sSlice = N * N;

    // Transformations Equations
    int sEq1 = (sSlice + sLine) / 2;
    int sEq2 = (sSlice - sLine) / 2;

    __syncthreads();

    // Thread Index (1D)
    int xThreadIdx = threadIdx.x;
    int yThreadIdx = threadIdx.y;

    __syncthreads();

    // Block Width & Height
    int blockWidth = blockDim.x;
    int blockHeight = blockDim.y;

    __syncthreads();

    // Thread Index (2D)
    int xIndex = blockIdx.x * blockWidth + xThreadIdx;
    int yIndex = blockIdx.y * blockHeight + yThreadIdx;

    __syncthreads();

    // Thread Index Converted into 1D Index
    int index = (yIndex * N) + xIndex;

    __syncthreads();

    if (xIndex < N / 2)
    {
        if (yIndex < N / 2)
        {
            // First Quad
            output[index] = input[index + sEq1];
            __syncthreads();
        }
        else
        {
            // Third Quad
            output[index] = input[index - sEq2];
            __syncthreads();
        }
    }
    else
    {
        if (yIndex < N / 2)
        {
            // Second Quad
            output[index] = input[index + sEq2];
            __syncthreads();
        }
        else
        {
            // Fourth Quad
            output[index] = input[index - sEq1];
            __syncthreads();
        }
    }
}

#endif // CUFFTSHIFT_2D_CU
