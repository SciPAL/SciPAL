#include "hip/hip_runtime.h"
/*********************************************************************
 * Copyright © 2011-2014,
 * Marwan Abdellah: <abdellah.marwan@gmail.com>
 *
 * This library (cufftShift) is free software; you can redistribute it
 * and/or modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 ********************************************************************/
 
#ifndef CUFFTSHIFT_3D_CU
#define CUFFTSHIFT_3D_CU

#include <cufftShiftShared.h>

template <typename T>
__global__
void cufftShift_3D_slice_kernel(T* input, T* output, int N, int zIndex)
{
    // 3D Volume & 2D Slice & 1D Line
    int sLine = N;
    int sSlice = N * N;
    int sVolume = N * N * N;

    // Transformations Equations
    int sEq1 = (sVolume + sSlice + sLine) / 2;
    int sEq2 = (sVolume + sSlice - sLine) / 2;
    int sEq3 = (sVolume - sSlice + sLine) / 2;
    int sEq4 = (sVolume - sSlice - sLine) / 2;

    // Thread
    int xThreadIdx = threadIdx.x;
    int yThreadIdx = threadIdx.y;

    // Block Width & Height
    int blockWidth = blockDim.x;
    int blockHeight = blockDim.y;

    // Thread Index 2D
    int xIndex = blockIdx.x * blockWidth + xThreadIdx;
    int yIndex = blockIdx.y * blockHeight + yThreadIdx;

    // Thread Index Converted into 1D Index
    int index = (zIndex * sSlice) + (yIndex * sLine) + xIndex;

    if (zIndex < N / 2)
    {
        if (xIndex < N / 2)
        {
            if (yIndex < N / 2)
            {
                // First Quad
                output[index] = input[index + sEq1];
            }
            else
            {
                // Third Quad
                output[index] = input[index + sEq3];
            }
        }
        else
        {
            if (yIndex < N / 2)
            {
                // Second Quad
                output[index] = input[index + sEq2];
            }
            else
            {
                // Fourth Quad
                output[index] = input[index + sEq4];
            }
        }
    }

    else
    {
        if (xIndex < N / 2)
        {
            if (yIndex < N / 2)
            {
                // First Quad
                output[index] = input[index - sEq4];
            }
            else
            {
                // Third Quad
                output[index] = input[index - sEq2];
            }
        }
        else
        {
            if (yIndex < N / 2)
            {
                // Second Quad
                output[index] = input[index - sEq3];
            }
            else
            {
                // Fourth Quad
                output[index] = input[index - sEq1];
            }
        }
    }
}

template <typename T>
void cufftShift_3D_kernel(T* input, T* output, int N, dim3 block, dim3 grid)
{
    for (int i = 0; i < N; i++)
        cufftShift_3D_slice_kernel <<< grid, block >>> (input, output, N, i);
}

template
void cufftShift_3D_kernel <hipfftReal>
(hipfftReal* input, hipfftReal* output, int N, dim3 block, dim3 grid);

template
void cufftShift_3D_kernel <hipfftDoubleReal>
(hipfftDoubleReal* input, hipfftDoubleReal* output, int N, dim3 block, dim3 grid);

template
void cufftShift_3D_kernel <hipfftComplex>
(hipfftComplex* input, hipfftComplex* output, int N, dim3 block, dim3 grid);

template
void cufftShift_3D_kernel <hipfftDoubleComplex>
(hipfftDoubleComplex* input, hipfftDoubleComplex* output, int N, dim3 block, dim3 grid);

#endif // CUFFTSHIFT_3D_CU
